 #include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiptensor.h>
#include<hip/hip_runtime.h>

#include <unordered_map>
#include <vector>

// Handle cuTENSOR errors
#define HANDLE_ERROR(x) {                                                              \
  const auto err = x;                                                                  \
  if( err != HIPTENSOR_STATUS_SUCCESS )                                                 \
  { printf("Error: %s in line %d\n", hiptensorGetErrorString(err), __LINE__); exit(-1); } \
}

void print(std::vector <int> const &a) {
   std::cout << "The vector elements are : ";

   for(int i=0; i < a.size(); i++)
   std::cout << a.at(i) << ' ';
}

int main(int argc, char** argv)
{
  // Host element type definition
  typedef float floatTypeA;
  typedef float floatTypeB;
  typedef float floatTypeC;
  typedef float floatTypeCompute;

  // CUDA types
  hipDataType typeA = HIP_R_32F;
  hipDataType typeB = HIP_R_32F;
  hipDataType typeC = HIP_R_32F;
  hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

  floatTypeCompute alpha = (floatTypeCompute)1.1f;
  floatTypeCompute beta  = (floatTypeCompute)0.9f;

  printf("Include headers and define data types\n");

  //=====================

  // Create vector of modes--ie, indices along each axis of tensor
  std::vector<int> modeC{'a','b','m','u','n','v'};
  print(modeC)
  std::vector<int> modeA{'a','b','m','h','k','n'};
  std::vector<int> modeB{'u','k','v','h'};
  int nmodeA = modeA.size();
  int nmodeB = modeB.size();
  int nmodeC = modeC.size();

  // Extents--size of each axis, ie the index runs from 0 to extent-1
  std::unordered_map<int, int64_t> extent;
  extent['m'] = 2;
  extent['n'] = 2;
  extent['u'] = 2;
  extent['v'] = 2;
  extent['h'] = 2;
  extent['k'] = 2;
  extent['a'] = 2;
  extent['b'] = 2;

  // Create a vector of extents for each tensor
  std::vector<int64_t> extentC;
  for(auto mode : modeC)
      extentC.push_back(extent[mode]);
  std::vector<int64_t> extentA;
  for(auto mode : modeA)
      extentA.push_back(extent[mode]);
  std::vector<int64_t> extentB;
  for(auto mode : modeB)
      extentB.push_back(extent[mode]);

  printf("Define modes and extents\n");

  // ============================

  // Number of elements of each tensor
  /*size_t elementsA = 1;
  for(auto mode : modeA)
      elementsA *= extent[mode];
  size_t elementsB = 1;
  for(auto mode : modeB)
      elementsB *= extent[mode];
  size_t elementsC = 1;
  for(auto mode : modeC)
      elementsC *= extent[mode];

  // Size in bytes
  size_t sizeA = sizeof(floatTypeA) * elementsA;
  size_t sizeB = sizeof(floatTypeB) * elementsB;
  size_t sizeC = sizeof(floatTypeC) * elementsC;

  // Allocate on device
  void *A_d, *B_d, *C_d;
  hipMalloc((void**)&A_d, sizeA);
  hipMalloc((void**)&B_d, sizeB);
  hipMalloc((void**)&C_d, sizeC);

  // Allocate on host
  floatTypeA *A = (floatTypeA*) malloc(sizeof(floatTypeA) * elementsA);
  floatTypeB *B = (floatTypeB*) malloc(sizeof(floatTypeB) * elementsB);
  floatTypeC *C = (floatTypeC*) malloc(sizeof(floatTypeC) * elementsC);

  // Initialize data on host
  for(int64_t i = 0; i < elementsA; i++)
      A[i] = (((float) rand())/RAND_MAX - 0.5)*100;
  for(int64_t i = 0; i < elementsB; i++)
      B[i] = (((float) rand())/RAND_MAX - 0.5)*100;
  for(int64_t i = 0; i < elementsC; i++)
      C[i] = (((float) rand())/RAND_MAX - 0.5)*100;

  // Copy to device
  hipMemcpy(C_d, C, sizeC, hipMemcpyHostToDevice);
  hipMemcpy(A_d, A, sizeA, hipMemcpyHostToDevice);
  hipMemcpy(B_d, B, sizeB, hipMemcpyHostToDevice);

  printf("Allocate, initialize and transfer tensors\n");

  // ==============================

  // Initialize cuTENSOR library
  hiptensorHandle_t handle;
  cutensorInit(&handle);

  // Create Tensor Descriptors
  hiptensorTensorDescriptor_t descA;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( &handle,
              &descA,
              nmodeA,
              extentA.data(),
              NULL,// stride
              typeA, HIPTENSOR_OP_IDENTITY ) );

  hiptensorTensorDescriptor_t descB;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( &handle,
              &descB,
              nmodeB,
              extentB.data(),
              NULL,//stride
              typeB, HIPTENSOR_OP_IDENTITY ) );

  hiptensorTensorDescriptor_t descC;
  HANDLE_ERROR( hiptensorInitTensorDescriptor( &handle,
              &descC,
              nmodeC,
              extentC.data(),
              NULL,//stride
              typeC, HIPTENSOR_OP_IDENTITY ) );

  printf("Initialize cuTENSOR and tensor descriptors\n");

  // ==========================

   //Retrieve the memory alignment for each tensor
   uint32_t alignmentRequirementA;
   HANDLE_ERROR( cutensorGetAlignmentRequirement( &handle,
              A_d,
              &descA,
              &alignmentRequirementA) );

   uint32_t alignmentRequirementB;
   HANDLE_ERROR( cutensorGetAlignmentRequirement( &handle,
              B_d,
              &descB,
              &alignmentRequirementB) );

   uint32_t alignmentRequirementC;
   HANDLE_ERROR( cutensorGetAlignmentRequirement( &handle,
              C_d,
              &descC,
              &alignmentRequirementC) );

  printf("Query best alignment requirement for our pointers\n");

  // ====================================

  // Create the Contraction Descriptor
  cutensorContractionDescriptor_t desc;
  HANDLE_ERROR( cutensorInitContractionDescriptor( &handle,
              &desc,
              &descA, modeA.data(), alignmentRequirementA,
              &descB, modeB.data(), alignmentRequirementB,
              &descC, modeC.data(), alignmentRequirementC,
              &descC, modeC.data(), alignmentRequirementC,
              typeCompute) );

  printf("Initialize contraction descriptor\n");

  // ==================================
  // Set the algorithm to use
  cutensorContractionFind_t find;
  HANDLE_ERROR( cutensorInitContractionFind(
              &handle, &find,
              HIPTENSOR_ALGO_DEFAULT) );

  printf("Initialize settings to find algorithm\n");

  // =================================

  // Query workspace
  size_t worksize = 0;
  HANDLE_ERROR( cutensorContractionGetWorkspace(&handle,
              &desc,
              &find,
              CUTENSOR_WORKSPACE_RECOMMENDED, &worksize ) );

  // Allocate workspace
  void *work = nullptr;
  if(worksize > 0)
  {
      if( hipSuccess != hipMalloc(&work, worksize) ) // This is optional!
      {
          work = nullptr;
          worksize = 0;
      }
  }

  printf("Query recommended workspace size and allocate it\n");

  // ===============================

  // Create Contraction Plan
  hiptensorContractionPlan_t plan;
  HANDLE_ERROR( cutensorInitContractionPlan(&handle,
                                            &plan,
                                            &desc,
                                            &find,
                                            worksize) );

  printf("Create plan for contraction\n");

  // ================================

  hiptensorStatus_t err;

  // Execute the tensor contraction
  err = hiptensorContraction(&handle,
                            &plan,
                     (void*)&alpha, A_d,
                                    B_d,
                     (void*)&beta,  C_d,
                                    C_d,
                            work, worksize, 0); // stream );
  hipDeviceSynchronize();

  // Check for errors
  if(err != HIPTENSOR_STATUS_SUCCESS)
  {
      printf("ERROR: %s\n", hiptensorGetErrorString(err));
  }

  printf("Execute contraction from plan\n");

  // ============================

  if ( A ) free( A );
  if ( B ) free( B );
  if ( C ) free( C );
  if ( A_d ) hipFree( A_d );
  if ( B_d ) hipFree( B_d );
  if ( C_d ) hipFree( C_d );
  if ( work ) hipFree( work );

  printf("Successful completion\n");

  return 0;
} 

/*#include <stdlib.h>
#include <stdio.h>

#include <hip/hip_runtime.h>
#include <hiptensor.h>

int main(int argc, char** argv)
{
  // Host element type definition
  typedef float floatTypeA;
  typedef float floatTypeB;
  typedef float floatTypeC;
  typedef float floatTypeCompute;

  // CUDA types
  hipDataType typeA = HIP_R_32F;
  hipDataType typeB = HIP_R_32F;
  hipDataType typeC = HIP_R_32F;
  hiptensorComputeDescriptor_t typeCompute = HIPTENSOR_COMPUTE_DESC_32F;

  floatTypeCompute alpha = (floatTypeCompute)1.1f;
  floatTypeCompute beta  = (floatTypeCompute)0.9f;

  printf("Include headers and define data types\n");

  return 0;
}*/
